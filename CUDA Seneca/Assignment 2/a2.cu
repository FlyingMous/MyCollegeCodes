#include "hip/hip_runtime.h"
/*
//****************************************************************************80
//
//  Purpose:
//
//    MAIN is the main program for HEATED_PLATE.
//
//  Discussion:
//
//    This code solves the steady state heat equation on a rectangular region.
//
//    The sequential version of this program needs approximately
//    18/epsilon iterations to complete.
//
//
//    The physical region, and the boundary conditions, are suggested
//    by this diagram;
//
//                   W = 0
//             +------------------+
//             |                  |
//    W = 100  |                  | W = 100
//             |                  |
//             +------------------+
//                   W = 100
//
//    The region is covered with a grid of M by N nodes, and an N by N
//    array W is used to record the temperature.  The correspondence between
//    array indices and locations in the region is suggested by giving the
//    indices of the four corners:
//
//                  I = 0
//          [0][0]-------------[0][N-1]
//             |                  |
//      J = 0  |                  |  J = N-1
//             |                  |
//        [M-1][0]-----------[M-1][N-1]
//                  I = M-1
//
//    The steady state solution to the discrete heat equation satisfies the
//    following condition at an interior grid point:
//
//      W[Central] = (1/4) * ( W[North] + W[South] + W[East] + W[West] )
//
//    where "Central" is the index of the grid point, "North" is the index
//    of its immediate neighbor to the "north", and so on.
//
//    Given an approximate solution of the steady state heat equation, a
//    "better" solution is given by replacing each interior point by the
//    average of its 4 neighbors - in other words, by using the condition
//    as an ASSIGNMENT statement:
//
//      W[Central]  <=  (1/4) * ( W[North] + W[South] + W[East] + W[West] )
//
//    If this process is repeated often enough, the difference between successive
//    estimates of the solution will go to zero.
//
//    This program carries out such an iteration, using a tolerance specified by
//    the user, and writes the final estimate of the solution to a file that can
//    be used for graphic processing.
//
//  Licensing:
//
//    This code is distributed under the GNU LGPL license.
//
//  Modified:
//
//    22 July 2008
//
//  Author:
//
//    Original C version by Michael Quinn.
//    C++ version by John Burkardt.
//
//  Reference:
//
//    Michael Quinn,
//    Parallel Programming in C with MPI and OpenMP,
//    McGraw-Hill, 2004,
//    ISBN13: 978-0071232654,
//    LC: QA76.73.C15.Q55.
//
//  Parameters:
//
//    Commandline argument 1, double EPSILON, the error tolerance.
//
//    Commandline argument 2, char *OUTPUT_FILENAME, the name of the file into which
//    the steady state solution is written when the program has completed.
//
//  Local parameters:
//
//    Local, double DIFF, the norm of the change in the solution from one iteration
//    to the next.
//
//    Local, double MEAN, the average of the boundary values, used to initialize
//    the values of the solution in the interior.
//
//    Local, double U[M][N], the solution at the previous iteration.
//
//    Local, double W[M][N], the solution computed at the latest iteration.
//
//****************************************************************************80
*/
# include <cstdlib>
# include <iostream>
# include <iomanip>
# include <fstream>
# include <cmath>
# include <ctime>
# include <string>
# include <hip/hip_runtime.h>

using namespace std;
# define M 1000
# define N 1000


/*double ctime;
double ctime1;
double ctime2;*/


//__device__ double *h_Diff;

ofstream output;
char output_filename[80];
FILE *fp;

int i;
int iterations;
int iterations_print;
int j;
int success;

const int ntpb = 32;

double mean;
double diff;
double epsilon;
double u[M*N];
double w[M*N];

void setBoundaryValue();
void setAverageBoundary();
void writeToFile();
void getHeat();
int main(int argc, char *argv[]);
double cpu_time();

__global__ void copyMat(const double *w, double *u){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < M && j < N) {
		u[i * M + j] = w[i * M + j];
	}
	__syncthreads();
}
__global__ void calcHeat(double *w, const double *u, double *d, int m, int n, double* d_array){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if ( i == 0 )i++; 
	if ( j == 0 )j++;
	if (i < m && j < n) {
		w[i * m + j] = (u[(i - 1) * m + j] + u[(i + 1) * m + j] + u[i * m + (j - 1)] + u[i * m + (j + 1)]) / 4.0;
		d_array[i * m + j] = w[i * m + j] - u[i * m + j];
		if( d_array[i * m + j] < 0 ){d_array[i * m + j] *= -1;}
	}
	*d = -1;
	__syncthreads();
}
__global__ void bigDiff(double* d_array, double* d, int m, int n){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int x = 1; i+x < m*n; x*=2) {
		if (d_array[i] > *d || d_array[i + x] > *d){
			if (d_array[i] > d_array[i + x])
				*d = d_array[i];
			else
				*d = d_array[i + x];
		}
		__syncthreads();
	}
}

void setBoundaryValue(){
	for (i = 1; i < M - 1; i++)
	{
		w[i*M] = 100.0;
	}
	for (i = 0; i < M - 1; i++)
	{
		w[i * M + N - 1] = 100.0;
	}
	for (j = 0; j < N; j++)
	{
		w[M - 1 + j] = 100.0;
	}
	for (j = 0; j < N; j++)
	{
		w[j] = 0.0;
	}
}
void setAverageBoundary(){
	mean = 0.0;
	for (i = 1; i < M - 1; i++)
	{
		mean = mean + w[i*M];
	}
	for (i = 1; i < M - 1; i++)
	{
		mean = mean + w[i * M + N - 1];
	}
	for (j = 0; j < N; j++)
	{
		mean = mean + w[M - 1 + j];
	}
	for (j = 0; j < N; j++)
	{
		mean = mean + w[j];
	}
	mean = mean / (double)(2 * M + 2 * N - 4);
	// 
	//  Initialize the interior solution to the mean value.
	//
	for (i = 1; i < M - 1; i++)
	{
		for (j = 1; j < N - 1; j++)
		{
			w[i * M + j] = mean;
		}
	}
}
void writeToFile(){
	/*
	output.open(output_filename);

	output << M << "\n";
	output << N << "\n";

	for (i = 0; i < M; i++)
	{
		for (j = 0; j < N; j++)
		{
			output << "  " << w[i * M + j];
		}
		output << "\n";
	}
	output.close();

	cout << "\n";
	cout << "  Solution written to the output file \"" << output_filename << "?\"?\n";
	*/
}

void getHeat(){
	double* h_u;
	double* h_w;
	double* h_d;
	double* h_Diff;
	double x = 0;

	hipMalloc((void**)&h_u, M * N * sizeof(double));
	hipMalloc((void**)&h_w, M * N * sizeof(double));
	hipMalloc((void**)&h_d, M * N * sizeof(double));
	hipMalloc((void**)&h_Diff, sizeof(double));

	hipMemcpy(h_w, w, M * N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(h_Diff, &x, sizeof(double), hipMemcpyHostToDevice);

	int nbx = (M + ntpb - 1) / ntpb;
	int nby = (N + ntpb - 1) / ntpb;
	dim3 dGrid(nbx, nby);
	dim3 dBlock(ntpb, ntpb);

	iterations = 0;
	iterations_print = 1;
	cout << "\n";
	cout << " Iteration  Change\n";
	cout << "\n";

	while (epsilon <= diff)
	{
		//
		//  Save the old solution in U.
		//
		/*
		for (i = 0; i < M; i++)
		{
		for (j = 0; j < N; j++)
		{
		u[i * M + j] = w[i * M + j];
		}
		}
		*/
		copyMat << <dGrid, dBlock >> >(h_w, h_u);
		//
		//  Determine the new estimate of the solution at the interior points.
		//  The new solution W is the average of north, south, east and west neighbors.
		//

		calcHeat << <dGrid, dBlock >> >(h_w, h_u, h_Diff, M, N, h_d);
		bigDiff << <1, dBlock >> >(h_d, h_Diff,M,N);
		hipMemcpy(&diff, h_Diff, sizeof(double), hipMemcpyDeviceToHost);

		//cout << diff;
		//cout << endl;

		/*
		for (i = 1; i < M - 1; i++)
		{
		for (j = 1; j < N - 1; j++)
		{

		w[i * M + j] = (u[i - 1][j] + u[i + 1][j] + u[i][j - 1] + u[i][j + 1]) / 4.0;

		if (diff < fabs(w[i * M + j] - u[i * M + j]))
		{
		diff = fabs(w[i * M + j] - u[i * M + j]);
		}
		}
		}*/

		iterations++;
		if (iterations == iterations_print)
		{
			cout << "  " << setw(8) << iterations
				<< "  " << diff << "\n";
			iterations_print = 2 * iterations_print;
		}
	}
	hipMemcpy(w, h_w, M * N * sizeof(double), hipMemcpyDeviceToHost);
	
	hipFree(h_w);
	hipFree(h_u);
	hipFree(h_d);
	hipFree(h_Diff);
	hipDeviceReset();
}


int main(int argc, char *argv[]){
/*
	cout << "\n";
	cout << "HEATED_PLATE\n";
	cout << "  C++ version\n";
	cout << "  A program to solve for the steady state temperature distribution\n";
	cout << "  over a rectangular plate.\n";
	cout << "\n";
	cout << "  Spatial grid of " << M << " by " << N << " points.\n";
	// 
	//  Read EPSILON from the command line or the user.
	//
	if (argc < 2)
	{
		cout << "\n";
		cout << "  Enter EPSILON, the error tolerance:\n";
		cin >> epsilon;
	}
	else
	{
		success = sscanf(argv[1], "%f", &epsilon);

		if (success != 1)
		{
			cout << "\n";
			cout << "HEATED_PLATE\n";
			cout << "  Error reading in the value of EPSILON.\n";
			return 1;
		}
	}

	cout << "\n";
	cout << "  The iteration will be repeated until the change is <= "
		<< epsilon << "\n";
*/
	epsilon = 0.5f;
	diff = epsilon;
	// 
	//  Read OUTPUT_FILE from the command line or the user.
	//
	/*
	if (argc < 3)
	{
		cout << "\n";
		cout << "  Enter OUTPUT_FILENAME, the name of the output file:\n";
		cin >> output_filename;
	}
	else
	{
		success = sscanf(argv[2], "%s", output_filename);

		if (success != 1)
		{
			cout << "\n";
			cout << "HEATED_PLATE\n";
			cout << "  Error reading in the value of OUTPUT_FILENAME.\n";
			return 1;
		}
	}

	cout << "\n";
	cout << "  The steady state solution will be written to \"?"
		<< output_filename << "\".\n";
		*/
	// 
	//  Set the boundary values, which don't change. 
	//
	setBoundaryValue();
	//
	//  Average the boundary values, to come up with a reasonable
	//  initial value for the interior.
	// 
	setAverageBoundary();
	//
	//  iterate until the  new solution W differs from the old solution U
	//  by no more than EPSILON.
	//
	//ctime1 = cpu_time();

	getHeat();

	//ctime2 = cpu_time();
	//ctime = ctime2 - ctime1;

	cout << "\n";
	cout << "  " << setw(8) << iterations
		<< "  " << diff << "\n";
	cout << "\n";
	cout << "  Error tolerance achieved.\n";
	//	cout << "  CPU time = " << ctime << "\n";
	// 
	//  Write the solution to the output file.
	//
	//writeToFile();
	// 
	//  Terminate.
	//
	cout << "\n";
	cout << "HEATED_PLATE:\n";
	cout << "  Normal end of execution.\n";

	return 0;

# undef M
# undef N
}
//****************************************************************************80

double cpu_time()

//****************************************************************************80
//
//  Purpose:
//
//    CPU_TIME returns the current reading on the CPU clock.
//
//  Licensing:
//
//    This code is distributed under the GNU LGPL license. 
//
//  Modified:
//
//    06 June 2005
//
//  Author:
//
//    John Burkardt
//
//  Parameters:
//
//    Output, double CPU_TIME, the current reading of the CPU clock, in seconds.
//
{
	double value;

	value = (double)clock() / (double)CLOCKS_PER_SEC;

	return value;
}