#include "hip/hip_runtime.h"
/*
//****************************************************************************80
//
//  Purpose:
//
//    MAIN is the main program for HEATED_PLATE.
//
//  Discussion:
//
//    This code solves the steady state heat equation on a rectangular region.
//
//    The sequential version of this program needs approximately
//    18/epsilon iterations to complete.
//
//
//    The physical region, and the boundary conditions, are suggested
//    by this diagram;
//
//                   W = 0
//             +------------------+
//             |                  |
//    W = 100  |                  | W = 100
//             |                  |
//             +------------------+
//                   W = 100
//
//    The region is covered with a grid of M by N nodes, and an N by N
//    array W is used to record the temperature.  The correspondence between
//    array indices and locations in the region is suggested by giving the
//    indices of the four corners:
//
//                  I = 0
//          [0][0]-------------[0][N-1]
//             |                  |
//      J = 0  |                  |  J = N-1
//             |                  |
//        [M-1][0]-----------[M-1][N-1]
//                  I = M-1
//
//    The steady state solution to the discrete heat equation satisfies the
//    following condition at an interior grid point:
//
//      W[Central] = (1/4) * ( W[North] + W[South] + W[East] + W[West] )
//
//    where "Central" is the index of the grid point, "North" is the index
//    of its immediate neighbor to the "north", and so on.
//
//    Given an approximate solution of the steady state heat equation, a
//    "better" solution is given by replacing each interior point by the
//    average of its 4 neighbors - in other words, by using the condition
//    as an ASSIGNMENT statement:
//
//      W[Central]  <=  (1/4) * ( W[North] + W[South] + W[East] + W[West] )
//
//    If this process is repeated often enough, the difference between successive
//    estimates of the solution will go to zero.
//
//    This program carries out such an iteration, using a tolerance specified by
//    the user, and writes the final estimate of the solution to a file that can
//    be used for graphic processing.
//
//  Licensing:
//
//    This code is distributed under the GNU LGPL license.
//
//  Modified:
//
//    22 July 2008
//
//  Author:
//
//    Original C version by Michael Quinn.
//    C++ version by John Burkardt.
//
//  Reference:
//
//    Michael Quinn,
//    Parallel Programming in C with MPI and OpenMP,
//    McGraw-Hill, 2004,
//    ISBN13: 978-0071232654,
//    LC: QA76.73.C15.Q55.
//
//  Parameters:
//
//    Commandline argument 1, float EPSILON, the error tolerance.
//
//    Commandline argument 2, char *OUTPUT_FILENAME, the name of the file into which
//    the steady state solution is written when the program has completed.
//
//  Local parameters:
//
//    Local, float DIFF, the norm of the change in the solution from one iteration
//    to the next.
//
//    Local, float MEAN, the average of the boundary values, used to initialize
//    the values of the solution in the interior.
//
//    Local, float U[M][N], the solution at the previous iteration.
//
//    Local, float W[M][N], the solution computed at the latest iteration.
//
//****************************************************************************80
*/
# include <cstdlib>
# include <iostream>
# include <iomanip>
# include <fstream>
# include <cmath>
# include <ctime>
# include <string>
# include <hip/hip_runtime.h>

using namespace std;
# define M 1000
# define N 1000


/*float ctime;
float ctime1;
float ctime2;*/


//__device__ float *h_Diff;

ofstream output;
char output_filename[80];
FILE *fp;

int i;
int iterations;
int iterations_print;
int j;
int success;

const int ntpb = 32;

float mean;
float diff;
float epsilon;
float u[M*N];
float w[M*N];

void setBoundaryValue();
void setAverageBoundary();
void writeToFile();
void getHeat();
int main(int argc, char *argv[]);
float cpu_time();

__global__ void copyMat(const float *w, float *u){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < M && j < N) {
		u[i * M + j] = w[i * M + j];
	}
	__syncthreads();
}
__global__ void calcHeat(float *w, float *u, float *d, int m, int n, float* d_array){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	__shared__ float s_u[ntpb][ntpb];
	__shared__ float s_w[ntpb][ntpb];
	__shared__ float s_dif[ntpb][ntpb];
	if (tx < ntpb && ty < ntpb) {
		s_w[ty][tx] = w[j * M + i];
		s_u[ty][tx] = w[j * M + i];
	}
	__syncthreads();

	if ( ( tx < (ntpb-1) && ty < (ntpb-1) ) && ( tx >0 && ty > 0 ) && ( i < M && j < N ) ) {
		s_w[ty][tx] = ( s_u[ty - 1][tx] + s_u[ty + 1][tx] + s_u[ty][tx - 1] + s_u[ty][tx + 1] ) / 4.0;

		s_dif[ty][tx] = fabsf(s_w[ty][tx] - s_u[ty][tx]);

		//if (s_dif[ty][tx] < 0){ s_dif[ty][tx] *= -1; }
	}
	__syncthreads();
	if (tx < ntpb && ty < ntpb) {
		w[j * M + i] = s_w[ty][tx];
		//u[j * M + i] = s_w[ty][tx];
		d_array[j * M + i] = s_dif[ty][tx];
	}
	__syncthreads();
}
__global__ void bigDiff(float* d_array, float* d, int m, int n){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int x = 1; i + x < m*n; x *= 2) {
		if (d_array[i] > *d || d_array[i + x] > *d){
			if (d_array[i] > d_array[i + x])
				*d = d_array[i];
			else
				*d = d_array[i + x];
		}
		__syncthreads();
	}
}

void setBoundaryValue(){
	for (i = 1; i < M - 1; i++)
	{
		w[i*M] = 100.0;
	}
	for (i = 0; i < M - 1; i++)
	{
		w[i * M + N - 1] = 100.0;
	}
	for (j = 0; j < N; j++)
	{
		w[M - 1 + j] = 100.0;
	}
	for (j = 0; j < N; j++)
	{
		w[j] = 0.0;
	}
}
void setAverageBoundary(){
	mean = 0.0;
	for (i = 1; i < M - 1; i++)
	{
		mean = mean + w[i*M];
	}
	for (i = 1; i < M - 1; i++)
	{
		mean = mean + w[i * M + N - 1];
	}
	for (j = 0; j < N; j++)
	{
		mean = mean + w[M - 1 + j];
	}
	for (j = 0; j < N; j++)
	{
		mean = mean + w[j];
	}
	mean = mean / (float)(2 * M + 2 * N - 4);
	// 
	//  Initialize the interior solution to the mean value.
	//
	for (i = 1; i < M - 1; i++)
	{
		for (j = 1; j < N - 1; j++)
		{
			w[i * M + j] = mean;
		}
	}
}
void writeToFile(){
	/*
	output.open(output_filename);

	output << M << "\n";
	output << N << "\n";

	for (i = 0; i < M; i++)
	{
	for (j = 0; j < N; j++)
	{
	output << "  " << w[i * M + j];
	}
	output << "\n";
	}
	output.close();

	cout << "\n";
	cout << "  Solution written to the output file \"" << output_filename << "?\"?\n";
	*/
}

void getHeat(){
	float* d_u; // old device matrix
	float* d_w; // new device matrix
	float* d_d; // device difference matrix
	float* d_Diff; // device difference value

	hipMalloc((void**)&d_u, M * N * sizeof(float));
	hipMalloc((void**)&d_w, M * N * sizeof(float));
	hipMalloc((void**)&d_d, M * N * sizeof(float));
	hipMalloc((void**)&d_Diff, sizeof(float));

	hipMemcpy(d_w, w, M * N * sizeof(float), hipMemcpyHostToDevice);

	int nbx = (M + ntpb - 1) / ntpb;
	int nby = (N + ntpb - 1) / ntpb;
	dim3 dGrid(nbx, nby);
	dim3 dBlock(ntpb, ntpb);

	iterations = 0;
	iterations_print = 1;
	cout << "\n";
	cout << " Iteration  Change\n";
	cout << "\n";

	while (epsilon <= diff)
	{
		//
		//  Save the old solution in U.
		//
		//  Determine the new estimate of the solution at the interior points.
		//  The new solution W is the average of north, south, east and west neighbors.
		//

		calcHeat << <dGrid, dBlock >> >(d_w, d_u, d_Diff, M, N, d_d);
		bigDiff << <1, dBlock >> >(d_d, d_Diff, M, N);
		hipMemcpy(&diff, d_Diff, sizeof(float), hipMemcpyDeviceToHost);

		iterations++;
		if (iterations == iterations_print)
		{
			cout << "  " << setw(8) << iterations
				<< "  " << diff << "\n";
			iterations_print = 2 * iterations_print;
		}
	}
	hipMemcpy(w, d_w, M * N * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_w);
	hipFree(d_u);
	hipFree(d_d);
	hipFree(d_Diff);
	hipDeviceReset();
}


int main(int argc, char *argv[]){
	epsilon = 0.001f;
	diff = epsilon;
	// 
	//  Set the boundary values, which don't change. 
	//
	setBoundaryValue();
	//
	//  Average the boundary values, to come up with a reasonable
	//  initial value for the interior.
	// 
	setAverageBoundary();
	//
	//  iterate until the  new solution W differs from the old solution U
	//  by no more than EPSILON.
	//
	//ctime1 = cpu_time();

	getHeat();

	//ctime2 = cpu_time();
	//ctime = ctime2 - ctime1;

	cout << "\n";
	cout << "  " << setw(8) << iterations
		<< "  " << diff << "\n";
	cout << "\n";
	cout << "  Error tolerance achieved.\n";
	//	cout << "  CPU time = " << ctime << "\n";
	// 
	//  Write the solution to the output file.
	//
	//writeToFile();
	// 
	//  Terminate.
	//
	cout << "\n";
	cout << "HEATED_PLATE:\n";
	cout << "  Normal end of execution.\n";

	return 0;

# undef M
# undef N
}
//****************************************************************************80

float cpu_time()

//****************************************************************************80
//
//  Purpose:
//
//    CPU_TIME returns the current reading on the CPU clock.
//
//  Licensing:
//
//    This code is distributed under the GNU LGPL license. 
//
//  Modified:
//
//    06 June 2005
//
//  Author:
//
//    John Burkardt
//
//  Parameters:
//
//    Output, float CPU_TIME, the current reading of the CPU clock, in seconds.
//
{
	float value;

	value = (float)clock() / (float)CLOCKS_PER_SEC;

	return value;
}